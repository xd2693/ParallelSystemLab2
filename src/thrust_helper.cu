#include "hip/hip_runtime.h"
#include "thrust_helper.cuh"
#include <iostream>
#include <set>

/*void get_label_thrust(thrust::device_vector<double> & input_vals, 
                      thrust::device_vector<double> & old_centers,
                      thrust::device_vector<int> & labels,
                      thrust::device_vector<int> & labels_for_reduce,
                      thrust::device_vector<double> & new_centers, 
                      thrust::device_vector<int> & n_points,
                      int    dims,
                      int    n_vals,
                      int    n_cluster)
{
    thrust::counting_iterator<int> first_point(0);
    thrust::counting_iterator<int> last_point = first_point + n_vals;

}*/

void get_label_thrust(thrust::device_vector<double> & input_vals, 
                      thrust::device_vector<double> & old_centers,
                      thrust::device_vector<double> & new_centers,
                      thrust::device_vector<int> & buffer,
                      thrust::device_vector<int> & labels,
                      thrust::device_vector<int> & labels_for_reduce,
                      thrust::device_vector<int> & n_points,
                      int    dims,
                      int    n_vals,
                      int    n_cluster)
{

    printf("Sizes %lu %lu %lu %lu %lu %lu %lu\n", input_vals.size(), old_centers.size(), new_centers.size(), buffer.size(), labels.size(), labels_for_reduce.size(), n_points.size());
    double* input_vals_p = thrust::raw_pointer_cast(input_vals.data());
    double* old_centers_p = thrust::raw_pointer_cast(old_centers.data());
    double* new_centers_p = thrust::raw_pointer_cast(new_centers.data());
    int* buffer_p = thrust::raw_pointer_cast(buffer.data());
    int* labels_p = thrust::raw_pointer_cast(labels.data());
    int* labels_reduce_p = thrust::raw_pointer_cast(labels_for_reduce.data());
    int* n_points_p = thrust::raw_pointer_cast(n_points.data());
    thrust::sequence(thrust::device, labels.begin(), labels.end(), 0);
    CentoidAssignFunctor functor(input_vals_p, old_centers_p, labels_p, labels_reduce_p, n_points_p, dims, n_cluster);
    thrust::for_each(thrust::device, labels.begin(), labels.end(), functor);
    
    int check_range = 1000;
    thrust::device_vector<int> label_check(labels_for_reduce.begin(), labels_for_reduce.begin()+check_range);
    thrust::device_vector<int> owner(n_points.begin(), n_points.end());
    int max_label = 0;
    int min_label = 0;
    std::set<int> test;
    printf("Centoids own ");
    for (int i = 0; i < owner.size()) {
        printf("%d ", owner[i]);
    }
    printf("\n");
    for (int i = 0; i < check_range; i++) {
        int temp = label_check[i];
        max_label = std::max(max_label, temp);
        min_label = std::min(min_label, temp);
        test.emplace(temp);
    }
    printf("Label range (%d-%d) with %lu labels\n", min_label, max_label, test.size());
    
    thrust::reduce_by_key(thrust::device, labels_reduce_p, labels_reduce_p+check_range, input_vals_p, buffer_p, new_centers_p);
    thrust::stable_sort_by_key(thrust::device, buffer_p, buffer_p+buffer.size(), new_centers_p, thrust::less<int>());
}