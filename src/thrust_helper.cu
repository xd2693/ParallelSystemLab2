#include "hip/hip_runtime.h"
#include "thrust_helper.cuh"
#include <iostream>
#include <set>

/*void get_label_thrust(thrust::device_vector<double> & input_vals, 
                      thrust::device_vector<double> & old_centers,
                      thrust::device_vector<int> & labels,
                      thrust::device_vector<int> & labels_for_reduce,
                      thrust::device_vector<double> & new_centers, 
                      thrust::device_vector<int> & n_points,
                      int    dims,
                      int    n_vals,
                      int    n_cluster)
{
    thrust::counting_iterator<int> first_point(0);
    thrust::counting_iterator<int> last_point = first_point + n_vals;

}*/

void get_label_thrust(thrust::device_vector<double> & input_vals, 
                      thrust::device_vector<double> & old_centers,
                      thrust::device_vector<double> & new_centers,
                      thrust::device_vector<int> & buffer,
                      thrust::device_vector<int> & labels,
                      thrust::device_vector<int> & labels_for_reduce,
                      thrust::device_vector<int> & n_points,
                      int    dims,
                      int    n_vals,
                      int    n_cluster)
{

    printf("Sizes %lu %lu %lu %lu %lu %lu %lu\n", input_vals.size(), old_centers.size(), new_centers.size(), buffer.size(), labels.size(), labels_for_reduce.size(), n_points.size());
    


    double* input_vals_p = thrust::raw_pointer_cast(input_vals.data());
    double* old_centers_p = thrust::raw_pointer_cast(old_centers.data());
    double* new_centers_p = thrust::raw_pointer_cast(new_centers.data());
    int* buffer_p = thrust::raw_pointer_cast(buffer.data());
    int* labels_p = thrust::raw_pointer_cast(labels.data());
    int* labels_reduce_p = thrust::raw_pointer_cast(labels_for_reduce.data());
    int* n_points_p = thrust::raw_pointer_cast(n_points.data());
    thrust::sequence(thrust::device, labels.begin(), labels.end(), 0);
    CentoidAssignFunctor functor(input_vals_p, old_centers_p, labels_p, labels_reduce_p, n_points_p, dims, n_cluster);
    
    thrust::device_vector<int> owner_before(n_points.begin(), n_points.end());
    printf("Centoids own before");
    for (int i = 0; i < owner_before.size(); i++) {
        printf("%d ", owner_before[i]);
    }
    printf("\n");

    thrust::device_vector<int> label_check_1(labels.begin(), labels.end());
    printf("label_check_1");
    for (int i = 0; i < label_check_1.size(); i++) {
        printf("%d ", label_check_1[i]);
    }
    printf("\n");

    thrust::device_vector<double> input_check(input_vals.begin(), input_vals.begin()+20);
    printf("Input check");
    for (int i = 0; i < input_check.size(); i++)
    {
        printf("%.5f ", input_check[i]);
    }
    printf("\n");

    thrust::device_vector<double> newc_check(new_centers.begin(), new_centers.begin()+20);
    printf("newc_check");
    for (int i = 0; i < input_check.size(); i++)
    {
        printf("%.5f ", newc_check[i]);
    }
    printf("\n");

    thrust::for_each(thrust::device, labels.begin(), labels.end(), functor);
    
    int check_range = 5000;
    thrust::device_vector<int> label_check(labels_for_reduce.begin(), labels_for_reduce.begin()+check_range);
    thrust::device_vector<int> owner(n_points.begin(), n_points.end());
    int max_label = 0;
    int min_label = 0;
    std::set<int> test;
    printf("Centoids own ");
    for (int i = 0; i < owner.size(); i++) {
        printf("%d ", owner[i]);
    }
    printf("\n");
    for (int i = 0; i < check_range; i++) {
        int temp = label_check[i];
        max_label = std::max(max_label, temp);
        min_label = std::min(min_label, temp);
        test.emplace(temp);
    }
    printf("Label range (%d-%d) with %lu labels\n", min_label, max_label, test.size());
    
    thrust::reduce_by_key(thrust::device, labels_reduce_p, labels_reduce_p+check_range, input_vals_p, buffer_p, new_centers_p);
    thrust::stable_sort_by_key(thrust::device, buffer_p, buffer_p+buffer.size(), new_centers_p, thrust::less<int>());
}